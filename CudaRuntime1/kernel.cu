#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#define NUM_THREADS 128
#define NUM_CHANNELS 1
// Random sample from the image for debug purposes.
#define DEBUG_IMG_IDX 20
// AdderNET kernel size
#define KERNEL_RADIUS 5

typedef unsigned char uint8_t;

#include "stb_image.h"
#include "stb_image_write.h"
#include <stdint.h>

hipError_t addernetCUDA(uint8_t* out_img, uint8_t* img, uint8_t* addernet_kernel, const int kernel_radius, const int img_width, const int img_length);

// Assuming VALID padding (size decreases)
// Index: kernel_radius -> img_width - kernel_radius - 1
// using square kernels like 5x5 (radius 2), 11x11 (radius 5)
// Single input & output channels
__global__ void addernetKernel(uint8_t* dev_out, uint8_t* dev_img, uint8_t* addernet_kernel, const int kernel_radius, const int img_width, const int img_length) {

    unsigned int tid_x = threadIdx.x;
    unsigned int tid_y = threadIdx.y;
    // Kernel radius can be added in the loop instead of using minus values.
    unsigned int idx_x = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    unsigned int idx_y = blockIdx.y * (blockDim.y * 2) + threadIdx.y;

    int accumulator = 0;
    int kernel_length = 2 * kernel_radius + 1; 

    for (size_t j = 0; j <= kernel_length; j++)
    {
        for (size_t i = 0; i <= kernel_length; i++)
        {
            // Read from global memory one by one. Shared memory can be used for optimization.
            if (idx_x + kernel_length < img_width && idx_y + kernel_length < img_length) {
                accumulator += abs(dev_img[(idx_y + j) * img_width + idx_x + i] - addernet_kernel[j * kernel_length + i]);
            }
        }
    }
    if (idx_x + kernel_radius < img_width && idx_y + kernel_radius < img_length) {
        dev_out[idx_x, idx_y] = accumulator;
    }
    __syncthreads();
}


int main()
{
    int width; //image width
    int height; //image height
    int bpp;  //bytes per pixel if the image was RGB (not used)

    // Load a grayscale bmp image to an unsigned integer array with its height and weight.
    //  (uint8_t is an alias for "unsigned char")
    uint8_t* image = stbi_load("./samples/640x426.bmp", &width, &height, &bpp, NUM_CHANNELS);

    // Print for sanity check
    printf("Bytes per pixel: %d \n", bpp / 3); //Image is grayscale, so bpp / 3;
    printf("Height: %d \n", height);
    printf("Width: %d \n", width);
    printf("Number of threads: %d \n", NUM_THREADS);

    // Fill flattened kernel (random with seed)
    const int kernel_radius = KERNEL_RADIUS;
    const int kernel_size = pow(2 * kernel_radius + 1, 2);
    uint8_t* addernet_kernel = (uint8_t*)malloc(kernel_size);
    srand(1);
    for (size_t i = 0; i < kernel_size; i++)
    {
       addernet_kernel[i] = rand() % 50;
    }

    // Initialize 2D output array
    uint8_t* out_image = (uint8_t*)malloc(width * height);

    // Get timing info
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    hipError_t cudaStatus = addernetCUDA(out_image, image, addernet_kernel, kernel_radius, width, height);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Execution took %3.5f ms \n", time);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "contrastEnhancementCuda failed!");
        return 1;
    }

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    // Write image array into a bmp file
    stbi_write_bmp("./out_img_640x426.bmp", width, height, 1, out_image);

    // Deallocate memory
    stbi_image_free(image);
    free(out_image);
    free(addernet_kernel);
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addernetCUDA(uint8_t* out_img, uint8_t* img, uint8_t* addernet_kernel, const int kernel_radius, const int img_width, const int img_length)
{
    int blockSize = NUM_THREADS;
    int gridSize = img_width * img_length / blockSize + (img_width * img_length % blockSize != 0);

    // Temp CPU array that hold min values of each block. We need half of the gridSize since 
    uint8_t* min_array;
    min_array = (uint8_t*)malloc(ceil(gridSize / 2) * sizeof(uint8_t));
    // Device memory pointers for image and block minima
    uint8_t* dev_img = 0;
    uint8_t* dev_out = 0;
    uint8_t* dev_addernet_kernel = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);

    // Allocate GPU memory for the image and minima of seperate blocks
    cudaStatus = hipMalloc((void**)&dev_img, img_width * img_length * sizeof(uint8_t));
    cudaStatus = hipMalloc((void**)&dev_out, (img_width - 2 * kernel_radius) * (img_length - 2 * kernel_radius) * sizeof(uint8_t));
    cudaStatus = hipMalloc((void**)&dev_addernet_kernel, pow(2 * kernel_radius + 1, 2) * sizeof(uint8_t));
    // Copy the image from host memory to GPU.
    hipMemcpy(dev_img, img, img_width * img_length * sizeof(uint8_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaMempcy img failed!");
        goto Error;
    }


    dim3 grid, block;
    block.x = blockSize;
    grid.x = gridSize;

    addernetKernel << <grid, block >> > (dev_out, dev_img, dev_addernet_kernel, kernel_radius, img_width, img_length);

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(out_img, dev_out, (img_width - 2 * kernel_radius) * (img_length - 2 * kernel_radius) * sizeof(uint8_t), hipMemcpyDeviceToHost);

    return cudaStatus;

Error:
    hipFree(dev_img);
    hipFree(dev_out);
    hipFree(dev_addernet_kernel);


    return cudaStatus;
}
