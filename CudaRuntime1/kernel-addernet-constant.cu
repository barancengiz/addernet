#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <cstdio>

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#define NUM_THREADS 1024
#define NUM_CHANNELS 1
// Random sample from the image for debug purposes.
#define DEBUG_IMG_IDX 20
// AdderNET kernel size
#define KERNEL_RADIUS 1
#define KERNEL_SIZE (2*KERNEL_RADIUS+1)*(2*KERNEL_RADIUS+1)

typedef unsigned char uint8_t;

#include "stb_image.h"
#include "stb_image_write.h"
#include <cstdint>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define GPUTHREADSIZE 1024

__constant__ uint8_t addernet_const_kernel[KERNEL_SIZE];

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
//    else{
//        fprintf(stderr, "GPUassertNOERROR: %s %s %d\n", hipGetErrorString(code), file, line);
//    }
}

hipError_t
addernetCUDA(uint8_t *out_img, uint8_t *img, uint8_t *addernet_kernel, int kernel_radius, int img_width,
             int img_height);

// Assuming VALID padding (size decreases)
// Index: kernel_radius -> img_width - kernel_radius - 1
// using square kernels like 5x5 (radius 2), 11x11 (radius 5)
// Single input & output channels
__global__ void
addernetKernel(uint8_t *dev_out, const uint8_t *dev_img, const int kernel_radius,
               const int img_width, const int imgHeigth) {

    unsigned int tid_x = threadIdx.x;
    unsigned int tid_y = threadIdx.y;
    // Kernel radius can be added in the loop instead of using minus values.
    unsigned int idx_x = blockIdx.x * (blockDim.x) + threadIdx.x;
    auto row = idx_x / img_width;
    auto col = idx_x % img_width;
//    unsigned int idx_y = blockIdx.y * (blockDim.y * 2) + threadIdx.y;

    int accumulator = 0;
    int kernel_length = 2 * kernel_radius + 1;

    for (size_t j = 0; j < kernel_length; j++) {
        for (size_t i = 0; i < kernel_length; i++) {
            // Read from global memory one by one. Shared memory can be used for optimization.
            if (col + kernel_length <= img_width && row + kernel_length <= imgHeigth) {
//                int i1 = -abs(dev_img[(row + j) * img_width + col + i] - addernet_const_kernel[j * kernel_length + i]);
//                uint8_t i2 = dev_img[(row + j) * img_width + col + i] * addernet_const_kernel[j * kernel_length + i];
                accumulator += -abs(dev_img[(row + j) * img_width + col + i] - addernet_const_kernel[j * kernel_length + i]);
//                accumulator += dev_img[(row + j) * img_width + col + i] * addernet_const_kernel[j * kernel_length + i];
            }
        }
    }
    if (col + kernel_length <= img_width && row + kernel_length <= imgHeigth) {
        dev_out[(row * (img_width - kernel_radius * 2)) + col] = accumulator;
    }
    __syncthreads();
}

// Matrix summation where each thread covers an element each
__global__ void mat_mean(const long *mean_vals, long *mean_outs, int tot_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= tot_size) return;

    //Copy memory
    extern __shared__ long means[];
    means[threadIdx.x] = mean_vals[idx];

    __syncthreads();

    //Reduce max

    int tid = threadIdx.x;
    if (blockDim.x > 1) {
        unsigned int lastDim = blockDim.x;
        for (unsigned int i = (blockDim.x + 1) / 2; i >= 1; (++i) >>= 1) {
            if (i == 1) {
                means[0] += means[1];
                __syncthreads();
                break;
            } else if (((lastDim & 0x1) == 1 && tid < (i - 1)) || (tid < i)) {
                means[tid] += means[tid + i];
            }
//            if (vec_idx < i-1 && i!=1) {
//                means[vec_idx] += means[vec_idx + i];
//            } else if (i == 1) {
//                means[0] += means[1];
//                __syncthreads();
//                break;
//            }
            (++lastDim) >>= 1;
            __syncthreads();
        }
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        mean_outs[blockIdx.x] = means[0];
    }

}

__global__ void square_sum(const uint8_t *mean_vals, long *mean_outs, long mean, int tot_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= tot_size) return;
    long diff = mean_vals[idx] - mean;
    mean_outs[idx] = diff * diff;
}

__global__ void normalize(const uint8_t *img, uint8_t *out_img, long mean, long var, int tot_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= tot_size) return;
    long diff = img[idx] - mean;
    out_img[idx] = diff / sqrtf(var);
}

int main() {
    int width; //image width
    int height; //image height
    int bpp;  //bytes per pixel if the image was RGB (not used)
    float total_time = 0;
    // Load a grayscale bmp image to an unsigned integer array with its height and weight.
    //  (uint8_t is an alias for "unsigned char")
    int num_runs = 10;
    for (int run = 0; run < num_runs; run++) {
        uint8_t *image = stbi_load("../CudaRuntime1/samples/640x426.bmp", &width, &height, &bpp, NUM_CHANNELS);

        // Print for sanity check
        printf("Bytes per pixel: %d \n", bpp / 3); //Image is grayscale, so bpp / 3;
        printf("Height: %d \n", height);
        printf("Width: %d \n", width);
        printf("Number of threads: %d \n", NUM_THREADS);

        // Fill flattened kernel (random with seed)
        const int kernel_radius = KERNEL_RADIUS;
        const int kernel_size = pow(2 * kernel_radius + 1, 2);
        auto *addernet_kernel = (uint8_t *) malloc(kernel_size);
        srand(1);
        for (size_t i = 0; i < kernel_size; i++) {
            if (i == kernel_size / 2) {
                addernet_kernel[i] = 1;
            } else {
                addernet_kernel[i] = 0;
            }
        }
//        addernet_kernel[0]=0;
//        addernet_kernel[1]=0.2;
//        addernet_kernel[2]=0.14;
//        addernet_kernel[3]=0.14;

        // Initialize 2D output array
        uint8_t *out_image = (uint8_t *) malloc((width - 2 * kernel_radius) * (height - 2 * kernel_radius));

        // Get timing info
        float time;
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start, 0);
        hipError_t cudaStatus = addernetCUDA(out_image, image, addernet_kernel, kernel_radius, width, height);

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        total_time += time;
        printf("Execution took %3.5f ms \n", time);

        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "contrastEnhancementCuda failed!");
            return 1;
        }

        cudaStatus = hipDeviceReset();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceReset failed!");
            return 1;
        }

        // Write image array into a bmp file
        stbi_write_bmp("./out_img_640x426.bmp", width - 2 * kernel_radius, height - 2 * kernel_radius, 1, out_image);

        // Deallocate memory
        stbi_image_free(image);
        free(out_image);
        free(addernet_kernel);
    }
    printf("Execution took %3.5f ms \n", total_time);
    printf("AVG: %3.5f ms \n", total_time / num_runs);

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t
addernetCUDA(uint8_t *out_img, uint8_t *img, uint8_t *addernet_kernel, const int kernel_radius, const int img_width,
             const int img_length) {
    int blockSize = NUM_THREADS;
    int gridSize = img_width * img_length / blockSize + (img_width * img_length % blockSize != 0);

    // Temp CPU array that hold min values of each block. We need half of the gridSize since
    uint8_t *min_array;
    min_array = (uint8_t *) malloc(ceil(gridSize / 2) * sizeof(uint8_t));
    // Device memory pointers for image and block minima
    uint8_t *dev_img;
    uint8_t *dev_out;
    uint8_t *dev_addernet_kernel;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
//    gpuErrchk(hipSetDevice(0));

    // Allocate GPU memory for the image and minima of seperate blocks
    gpuErrchk(hipMalloc((void **) &dev_img, img_width * img_length * sizeof(uint8_t)))
    int width = img_width - 2 * kernel_radius;
    int height = img_length - 2 * kernel_radius;
    gpuErrchk(hipMalloc((void **) &dev_out,
                         width * height * sizeof(uint8_t)))

    // Initialize constant memory in GPU
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(addernet_const_kernel), addernet_kernel, sizeof(uint8_t) * KERNEL_SIZE));

    // Copy the image from host memory to GPU.
    gpuErrchk(hipMemcpy(dev_img, img, img_width * img_length * sizeof(uint8_t), hipMemcpyHostToDevice));


    dim3 grid, block;
    block.x = blockSize;
    grid.x = gridSize;

    addernetKernel <<<grid, block >>>(dev_out, dev_img, kernel_radius, img_width, img_length);
    gpuErrchk(hipGetLastError())
    /* int n_blocks = 0;
     long *mean_h, *var_h;
     long *mean_d, *var_d;


     size_t tot_size_1 = width * height * sizeof(long);
     hipMalloc((void **) &mean_d, tot_size_1);
     hipMalloc((void **) &var_d, tot_size_1);
     hipMemset(mean_d, 0, tot_size_1);
     hipMemset(var_d, 0, tot_size_1);
     for (int i = 0; i < width * height; i++) {
         hipMemcpy(mean_d + i, dev_out, 1, hipMemcpyDeviceToDevice);
         hipMemcpy(var_d + i, dev_out, 1, hipMemcpyDeviceToDevice);
     }
     mean_h = (long *) malloc(sizeof(long));
     var_h = (long *) malloc(sizeof(long));
     int N = height * width;
     do {
         long *mean_temp_d;

         int block_size = min(GPUTHREADSIZE, N);
         n_blocks = N / block_size + (N % block_size == 0 ? 0 : 1);
         size_t count = n_blocks * sizeof(long);
         gpuErrchk(hipMalloc((void **) &mean_temp_d, count));
 //        gpuErrchk(hipMalloc((void **) &min_temp_d, count));
         gpuErrchk(hipMemset(mean_temp_d, 0, count));
 //        hipMemset(min_temp_d, 255, count);
         size_t i = block_size * sizeof(long);
         mat_mean<<< n_blocks, block_size, i>>>(mean_d, mean_temp_d, N);
         gpuErrchk(hipGetLastError())
 //        mat_min <<< n_blocks, block_size, block_size >>>(min_d, min_temp_d, N);
 //        err = hipGetLastError();
 //        gpuErrchk(err)
         hipDeviceSynchronize();
 //        gpuErrchk(hipMemcpy(min_d, min_temp_d, count, hipMemcpyDeviceToDevice));
         gpuErrchk(hipMemcpy(mean_d, mean_temp_d, count, hipMemcpyDeviceToDevice));
         hipDeviceSynchronize();

 //        gpuErrchk(hipFree(min_temp_d));
         gpuErrchk(hipFree(mean_temp_d));
         N = n_blocks;
     } while (n_blocks != 1);
     hipMemcpy(mean_h, mean_d, sizeof(long), hipMemcpyDeviceToHost);
 //    hipMemcpy(var_h, min_d, sizeof(long), hipMemcpyDeviceToHost);
     N = width * height;
     *mean_h = *mean_h / N;
     printf("mean: %d\n", *mean_h);
     int block_size = min(GPUTHREADSIZE, N);
     n_blocks = N / block_size + (N % block_size == 0 ? 0 : 1);
     square_sum<<< n_blocks, block_size>>>(dev_out, var_d, *mean_h, N);

     do {
         long *mean_temp_d;

         block_size = min(GPUTHREADSIZE, N);
         n_blocks = N / block_size + (N % block_size == 0 ? 0 : 1);
         size_t count = n_blocks * sizeof(long);
         gpuErrchk(hipMalloc((void **) &mean_temp_d, count));
 //        gpuErrchk(hipMalloc((void **) &min_temp_d, count));
         gpuErrchk(hipMemset(mean_temp_d, 0, count));
 //        hipMemset(min_temp_d, 255, count);
         size_t i = block_size * sizeof(long);
         mat_mean<<< n_blocks, block_size, i>>>(var_d, mean_temp_d, N);
         gpuErrchk(hipGetLastError())
 //        mat_min <<< n_blocks, block_size, block_size >>>(min_d, min_temp_d, N);
 //        err = hipGetLastError();
 //        gpuErrchk(err)
         hipDeviceSynchronize();
 //        gpuErrchk(hipMemcpy(min_d, min_temp_d, count, hipMemcpyDeviceToDevice));
         gpuErrchk(hipMemcpy(var_d, mean_temp_d, count, hipMemcpyDeviceToDevice));
         hipDeviceSynchronize();

 //        gpuErrchk(hipFree(min_temp_d));
         gpuErrchk(hipFree(mean_temp_d));
         N = n_blocks;
     } while (n_blocks != 1);
     hipMemcpy(var_h, var_d, sizeof(long), hipMemcpyDeviceToHost);
 //    hipMemcpy(var_h, min_d, sizeof(long), hipMemcpyDeviceToHost);
     N = width * height;
     printf("var: %d\n", *var_h);
     *var_h = *var_h / N;
     printf("var: %d\n", *var_h);

     N = width * height;
     block_size = min(GPUTHREADSIZE, N);
     n_blocks = N / block_size + (N % block_size == 0 ? 0 : 1);
     normalize<<< n_blocks, block_size>>>(dev_out, dev_out, *mean_h, *var_h, N);*/

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(out_img, dev_out,
                            width * height * sizeof(uint8_t),
                            hipMemcpyDeviceToHost);

    return cudaStatus;
}
