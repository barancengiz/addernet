#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <cstdio>

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#define NUM_THREADS 1024
#define NUM_CHANNELS 1
// Random sample from the image for debug purposes.
#define DEBUG_IMG_IDX 20
// AdderNET kernel size
#define KERNEL_RADIUS 5
#define KERNEL_SIZE (2*KERNEL_RADIUS+1)*(2*KERNEL_RADIUS+1)   // number of elements in the kernel

typedef unsigned char uint8_t;

#include "stb_image.h"
#include "stb_image_write.h"
#include <cstdint>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define GPUTHREADSIZE 1024

__constant__ uint8_t addernet_const_kernel[KERNEL_SIZE];

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
//    else{
//        fprintf(stderr, "GPUassertNOERROR: %s %s %d\n", hipGetErrorString(code), file, line);
//    }
}

hipError_t
addernetCUDA(uint8_t *out_img, uint8_t *img, uint8_t *addernet_kernel, int kernel_radius, int img_width,
             int img_height);

// Assuming VALID padding (size decreases)
// Index: kernel_radius -> img_width - kernel_radius - 1
// using square kernels like 5x5 (radius 2), 11x11 (radius 5)
// Single input & output channels
__global__ void
addernetKernel(uint8_t *dev_out, hipTextureObject_t dev_img, const int kernel_radius,
               const int img_width, const int imgHeigth) {

    unsigned int tid_x = threadIdx.x;
    unsigned int tid_y = threadIdx.y;
    // Kernel radius can be added in the loop instead of using minus values.
    unsigned int idx_x = blockIdx.x * (blockDim.x) + threadIdx.x;
    auto row = idx_x / img_width;
    auto col = idx_x % img_width;
//    unsigned int idx_y = blockIdx.y * (blockDim.y * 2) + threadIdx.y;

    int accumulator = 0;
    int kernel_length = 2 * kernel_radius + 1;

    for (size_t j = 0; j < kernel_length; j++) {
        for (size_t i = 0; i < kernel_length; i++) {
            // Read from global memory one by one. Shared memory can be used for optimization.
            if (col + kernel_length <= img_width && row + kernel_length <= imgHeigth) {
//                int i1 = -abs(tex2D<uint8_t>(dev_img, col + i, row + j) - addernet_const_kernel[j * kernel_length + i]);
//                uint8_t i2 = tex2D<uint8_t>(dev_img, col + i, row + j) * addernet_const_kernel[j * kernel_length + i]
//                accumulator += -abs(tex2D<uint8_t>(dev_img, col + i, row + j) - addernet_const_kernel[j * kernel_length + i]);
                accumulator += tex2D<uint8_t>(dev_img, col + i, row + j) * addernet_const_kernel[j * kernel_length + i];
            }
        }
    }
    if (col + kernel_length <= img_width && row + kernel_length <= imgHeigth) {
        dev_out[(row * (img_width - kernel_radius * 2)) + col] = accumulator;
    }
    __syncthreads();
}

// Matrix summation where each thread covers an element each
__global__ void mat_mean(const long *mean_vals, long *mean_outs, int tot_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= tot_size) return;

    //Copy memory
    extern __shared__ long means[];
    means[threadIdx.x] = mean_vals[idx];

    __syncthreads();

    //Reduce max

    int tid = threadIdx.x;
    if (blockDim.x > 1) {
        unsigned int lastDim = blockDim.x;
        for (unsigned int i = (blockDim.x + 1) / 2; i >= 1; (++i) >>= 1) {
            if (i == 1) {
                means[0] += means[1];
                __syncthreads();
                break;
            } else if (((lastDim & 0x1) == 1 && tid < (i - 1)) || (tid < i)) {
                means[tid] += means[tid + i];
            }
//            if (vec_idx < i-1 && i!=1) {
//                means[vec_idx] += means[vec_idx + i];
//            } else if (i == 1) {
//                means[0] += means[1];
//                __syncthreads();
//                break;
//            }
            (++lastDim) >>= 1;
            __syncthreads();
        }
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        mean_outs[blockIdx.x] = means[0];
    }

}

__global__ void square_sum(const uint8_t *mean_vals, long *mean_outs, long mean, int tot_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= tot_size) return;
    long diff = mean_vals[idx] - mean;
    mean_outs[idx] = diff * diff;
}

__global__ void normalize(const uint8_t *img, uint8_t *out_img, long mean, long var, int tot_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= tot_size) return;
    long diff = img[idx] - mean;
    out_img[idx] = diff / sqrtf(float(var));
}


int main() {
    int width; //image width
    int height; //image height
    int bpp;  //bytes per pixel if the image was RGB (not used)
    float total_time = 0;
    // Load a grayscale bmp image to an unsigned integer array with its height and weight.
    //  (uint8_t is an alias for "unsigned char")
    int num_of_runs = 10;
    for (int run = 0; run < num_of_runs; run++) {
        uint8_t *image = stbi_load("../CudaRuntime1/samples/5184x3456.bmp", &width, &height, &bpp, NUM_CHANNELS);

        // Print for sanity check
        printf("Bytes per pixel: %d \n", bpp / 3); //Image is grayscale, so bpp / 3;
        printf("Height: %d \n", height);
        printf("Width: %d \n", width);
        printf("Number of threads: %d \n", NUM_THREADS);

        // Fill flattened kernel (random with seed)
        const int kernel_radius = KERNEL_RADIUS;
        const int kernel_size = pow(2 * kernel_radius + 1, 2);
        auto *addernet_kernel = (uint8_t *) malloc(kernel_size);
        srand(1);
        for (size_t i = 0; i < kernel_size; i++) {
            if (i == kernel_size / 2) {
                addernet_kernel[i] = 1;
            } else {
                addernet_kernel[i] = 0;
            }
        }
//        addernet_kernel[0]=0;
//        addernet_kernel[1]=0.2;
//        addernet_kernel[2]=0.14;
//        addernet_kernel[3]=0.14;

        // Initialize 2D output array
        uint8_t *out_image = (uint8_t *) malloc((width - 2 * kernel_radius) * (height - 2 * kernel_radius));

        // Get timing info
        float time;
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start, 0);
        hipError_t cudaStatus = addernetCUDA(out_image, image, addernet_kernel, kernel_radius, width, height);

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        total_time += time;
        printf("Execution took %3.5f ms \n", time);

        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "contrastEnhancementCuda failed!");
            return 1;
        }

        cudaStatus = hipDeviceReset();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceReset failed!");
            return 1;
        }

        // Write image array into a bmp file
        stbi_write_bmp("./out_img_5184x3456.bmp", width - 2 * kernel_radius, height - 2 * kernel_radius, 1, out_image);

        // Deallocate memory
        stbi_image_free(image);
        free(out_image);
        free(addernet_kernel);
    }
    printf("Execution took %3.5f ms \n", total_time);
    printf("AVG: %3.5f ms \n", total_time / num_of_runs);

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t
addernetCUDA(uint8_t *out_img, uint8_t *img, uint8_t *addernet_kernel, const int kernel_radius, const int img_width,
             const int img_length) {
    int blockSize = NUM_THREADS;
    int gridSize = img_width * img_length / blockSize + (img_width * img_length % blockSize != 0);

    // Device memory pointers for image
    uint8_t *dev_out;
    uint8_t *dev_addernet_kernel;
    hipError_t cudaStatus;
    hipArray_t cuArray;

    // Choose which GPU to run on, change this on a multi-GPU system.
//    gpuErrchk(hipSetDevice(0));

    // CUDA Channel Descriptor
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);

    // Allocate CUDA array in device memory
    gpuErrchk(hipMallocArray(&cuArray, &channelDesc, img_width, img_length));

    // Set pitch of the source (the width in memory in bytes of the 2D array pointed to by src
    const size_t spitch = img_width * sizeof(uint8_t);

    // Specify texture
    hipResourceDesc resDesc{};
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    // Specify texture object parameters
    hipTextureDesc texDesc{};
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;

    // Create texture object
    hipTextureObject_t texObj = 0;
    gpuErrchk(hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL));

    // Copy data from host memory to CUDA array.
    gpuErrchk(hipMemcpy2DToArray(cuArray, 0, 0, img, spitch, img_width * sizeof(uint8_t), img_length,
                                  hipMemcpyHostToDevice));

    // Allocate GPU memory for the output
    int width = img_width - 2 * kernel_radius;
    int height = img_length - 2 * kernel_radius;
    gpuErrchk(hipMalloc((void **) &dev_out,
                         width * height * sizeof(uint8_t)))

    // Initialize constant memory in GPU
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(addernet_const_kernel), addernet_kernel, sizeof(uint8_t) * KERNEL_SIZE));

    dim3 grid, block;
    block.x = blockSize;
    grid.x = gridSize;

    addernetKernel <<<grid, block >>>(dev_out, texObj, kernel_radius, img_width, img_length);
    gpuErrchk(hipGetLastError())

//    int n_blocks = 0;
//    long *mean_h, *var_h;
//    long *mean_d, *var_d;
//
//
//    size_t tot_size_1 = width * height * sizeof(long);
//    hipMalloc((void **) &mean_d, tot_size_1);
//    hipMalloc((void **) &var_d, tot_size_1);
//    hipMemset(mean_d, 0, tot_size_1);
//    hipMemset(var_d, 0, tot_size_1);
//    for (int i = 0; i < width * height; i++) {
//        hipMemcpy(mean_d + i, dev_out, 1, hipMemcpyDeviceToDevice);
//        hipMemcpy(var_d + i, dev_out, 1, hipMemcpyDeviceToDevice);
//    }
//    mean_h = (long *) malloc(sizeof(long));
//    var_h = (long *) malloc(sizeof(long));
//    int N = height * width;
//    do {
//        long *mean_temp_d;
//
//        int block_size = min(GPUTHREADSIZE, N);
//        n_blocks = N / block_size + (N % block_size == 0 ? 0 : 1);
//        size_t count = n_blocks * sizeof(long);
//        gpuErrchk(hipMalloc((void **) &mean_temp_d, count));
////        gpuErrchk(hipMalloc((void **) &min_temp_d, count));
//        gpuErrchk(hipMemset(mean_temp_d, 0, count));
////        hipMemset(min_temp_d, 255, count);
//        size_t i = block_size * sizeof(long);
//        mat_mean<<< n_blocks, block_size, i>>>(mean_d, mean_temp_d, N);
//        gpuErrchk(hipGetLastError())
////        mat_min <<< n_blocks, block_size, block_size >>>(min_d, min_temp_d, N);
////        err = hipGetLastError();
////        gpuErrchk(err)
//        hipDeviceSynchronize();
////        gpuErrchk(hipMemcpy(min_d, min_temp_d, count, hipMemcpyDeviceToDevice));
//        gpuErrchk(hipMemcpy(mean_d, mean_temp_d, count, hipMemcpyDeviceToDevice));
//        hipDeviceSynchronize();
//
////        gpuErrchk(hipFree(min_temp_d));
//        gpuErrchk(hipFree(mean_temp_d));
//        N = n_blocks;
//    } while (n_blocks != 1);
//    hipMemcpy(mean_h, mean_d, sizeof(long), hipMemcpyDeviceToHost);
////    hipMemcpy(var_h, min_d, sizeof(long), hipMemcpyDeviceToHost);
//    N = width * height;
//    *mean_h = *mean_h / N;
//    printf("mean: %d\n", *mean_h);
//    int block_size = min(GPUTHREADSIZE, N);
//    n_blocks = N / block_size + (N % block_size == 0 ? 0 : 1);
//    square_sum<<< n_blocks, block_size>>>(dev_out, var_d, *mean_h, N);
//
//    do {
//        long *mean_temp_d;
//
//        block_size = min(GPUTHREADSIZE, N);
//        n_blocks = N / block_size + (N % block_size == 0 ? 0 : 1);
//        size_t count = n_blocks * sizeof(long);
//        gpuErrchk(hipMalloc((void **) &mean_temp_d, count));
////        gpuErrchk(hipMalloc((void **) &min_temp_d, count));
//        gpuErrchk(hipMemset(mean_temp_d, 0, count));
////        hipMemset(min_temp_d, 255, count);
//        size_t i = block_size * sizeof(long);
//        mat_mean<<< n_blocks, block_size, i>>>(var_d, mean_temp_d, N);
//        gpuErrchk(hipGetLastError())
////        mat_min <<< n_blocks, block_size, block_size >>>(min_d, min_temp_d, N);
////        err = hipGetLastError();
////        gpuErrchk(err)
//        hipDeviceSynchronize();
////        gpuErrchk(hipMemcpy(min_d, min_temp_d, count, hipMemcpyDeviceToDevice));
//        gpuErrchk(hipMemcpy(var_d, mean_temp_d, count, hipMemcpyDeviceToDevice));
//        hipDeviceSynchronize();
//
////        gpuErrchk(hipFree(min_temp_d));
//        gpuErrchk(hipFree(mean_temp_d));
//        N = n_blocks;
//    } while (n_blocks != 1);
//    hipMemcpy(var_h, var_d, sizeof(long), hipMemcpyDeviceToHost);
////    hipMemcpy(var_h, min_d, sizeof(long), hipMemcpyDeviceToHost);
//    N = width * height;
//    printf("var: %d\n", *var_h);
//    *var_h = *var_h / N;
//    printf("var: %d\n", *var_h);
//
//    N = width * height;
//    block_size = min(GPUTHREADSIZE, N);
//    n_blocks = N / block_size + (N % block_size == 0 ? 0 : 1);
//    normalize<<< n_blocks, block_size>>>(dev_out, dev_out, *mean_h, *var_h, N);

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(out_img, dev_out,
                            width * height * sizeof(uint8_t),
                            hipMemcpyDeviceToHost);

    // Destroy texture object
    gpuErrchk(hipDestroyTextureObject(texObj));
    gpuErrchk(hipFree(dev_out));

    return cudaStatus;
}
